
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<string.h>

__global__ void innerLoops(int n, int k, long* d_adj) {
    int i = blockIdx.x;
    int j = blockIdx.y;

    __syncthreads();

    long s = d_adj[i*n+k] + d_adj[k*n+j];                //[i][k]  [k][j]
    
    if(s < d_adj[i*n+j])                                 //[i][j]
        d_adj[i*n+j] = s;
}

__global__ void setMax(long* d_adj, int n){
    int x = threadIdx.x;
    int y = threadIdx.y;
    int pos = (x * n) + y;
    if(x == y)
        d_adj[pos] = 0;                                 //Diagonal elements
    else
        d_adj[pos] = __INT_MAX__;                       //Others
}

int main(int argc, char** argv){
    int i, j, k, n;
    printf("Enter the number of vertices : \n");
    scanf("%d", &n);

    long h_adj[n * n];
    long* d_adj;
    hipMalloc((void**)&d_adj, n * n * sizeof(long*));

    setMax<<<1, dim3(n, n, 1)>>>(d_adj, n); 

    hipMemcpy(h_adj, d_adj, n * n * sizeof(long), hipMemcpyDeviceToHost);
    
    hipFree(d_adj);

    while(1){
        printf("Click 1 to enter edge and 0 to finish.\n");
        scanf("%d", &k);
        if(!k)
            break;
        int s, d, w;
        printf("Enter start and end of edge in 1-ordering : \n");
        scanf("%d %d", &s, &d);
        if(s == d){
            printf("Invalid edge.\n");
            continue;
        }
        if(s > n || s < 1 || d > n || d < 1){
            printf("Invalid edge.\n");
            continue;
        }
        printf("Enter edge weight : \n");
        scanf("%d", &w);
        if(w < 0){
            printf("Invalid edge weight.\n");
            continue;
        }
        int pos = ((s - 1) * n) + (d - 1);
        h_adj[pos] = w;
    }
    
    hipDeviceSynchronize();

    for(k = 0; k < n; k++){
        innerLoops<<< dim3(1,1,1), dim3(n,n,1) >>>(n,k,d_adj);
    }

    for(i = 0;i < n; i++){
        for(j = 0;j < n; j++){
            int pos = (i * n) + j;    
            printf("%ld ", h_adj[pos]);
        }
        printf("\n");
    }

    return 0;
}