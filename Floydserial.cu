
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<string.h>

int main(){
    int i, j, k, n;
    printf("Enter the number of vertices : \n");
    scanf("%d", &n);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    long adj[n][n];
    for(i = 0;i < n; i++){
        for(j = 0;j < n; j++)
            adj[i][j] = __INT_MAX__;
    }
    while(1){
        printf("Click 1 to enter edge and 0 to finish.\n");
        scanf("%d", &k);
        if(!k)
            break;
        int s, d, w;
        printf("Enter start and end of edge in 1-ordering : \n");
        scanf("%d %d", &s, &d);
        if(s == d){
            printf("Invalid edge.\n");
            continue;
        }
        if(s > n || s < 1 || d > n || d < 1){
            printf("Invalid edge.\n");
            continue;
        }
        printf("Enter edge weight : \n");
        scanf("%d", &w);
        if(w < 0){
            printf("Invalid edge weight.\n");
            continue;
        }
        adj[s - 1][d - 1] = w;
    }
    for(i = 0; i < n; i++)
        adj[i][i] = 0;

    hipDeviceSynchronize();    
    hipEventRecord(start);

    for(k = 0; k < n; k++){
        for(i = 0; i < n; i++){
            for(j = 0;j < n; j++){
                long s = (long)adj[i][k] + (long)adj[k][j];
                if(s < adj[i][j])
                    adj[i][j] = s;
            }
        }
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    for(i = 0;i < n; i++){
        for(j = 0;j < n; j++)
            printf("%ld ", adj[i][j]);
        printf("\n");
    }
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("%f ms\n",milliseconds) ;

    return 0;
}