#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<string.h>
#include<hip/hip_runtime.h>

__global__ void innerLoops(int n, int k, long** d_adj) {
    int i=blockIdx.x;
    int j=blockIdx.y;
    
    long t1=*(*(d_adj+i)+k);
    long t2=*(*(d_adj+i)+j);
    long s = t1+t2;
    __syncthreads();

    long t3=*(*(d_adj+i)+j);
    if(s < t3)
        *(*(d_adj+i)+j) = s;
}

int main(){
    int i, j, k, n;
    printf("Enter the number of vertices : \n");
    scanf("%d", &n);
    long h_adj[n][n];
    for(i = 0;i < n; i++){
        for(j = 0;j < n; j++)
            h_adj[i][j] = __INT_MAX__;
    }
    while(1){
        printf("Click 1 to enter edge and 0 to finish.\n");
        scanf("%d", &k);
        if(!k)
            break;
        int s, d, w;
        printf("Enter start and end of edge in 1-ordering : \n");
        scanf("%d %d", &s, &d);
        if(s == d){
            printf("Invalid edge.\n");
            continue;
        }
        if(s > n || s < 1 || d > n || d < 1){
            printf("Invalid edge.\n");
            continue;
        }
        printf("Enter edge weight : \n");
        scanf("%d", &w);
        if(w < 0){
            printf("Invalid edge weight.\n");
            continue;
        }
        h_adj[s - 1][d - 1] = w;
    }
    for(i = 0; i < n; i++)
        h_adj[i][i] = 0;
    
    long** d_adj;
    hipMalloc((void**) &d_adj, n*sizeof(long*));
    for(int i=0; i<n; i++){
        hipMalloc(&d_adj[i], n*sizeof(long)); 
    }
    hipMemcpy(d_adj, h_adj, n*sizeof(long*), hipMemcpyHostToDevice);



    hipDeviceSynchronize();

    for(k = 0; k < n; k++){
        innerLoops<<< dim3(1,1,1), dim3(n,n,1) >>>(n,k,d_adj);
    }

    hipMemcpy(h_adj,d_adj,n*n,hipMemcpyDeviceToHost);

    for(i = 0;i < n; i++){
        for(j = 0;j < n; j++)
            printf("%ld ", h_adj[i][j]);
        printf("\n");
    }

    hipFree(d_adj);

    return 0;
}