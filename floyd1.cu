
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<string.h>

__global__ void setMax(long* d_adj, int n){
    int x = threadIdx.x;
    int y = threadIdx.y;
    int pos = (x * n) + y;
    if(x == y)
        d_adj[pos] = 0;                                 //Diagonal elements
    else
        d_adj[pos] = __INT_MAX__;                       //Others
}

int main(int argc, char** argv){
    int i, j, k, n;
    printf("Enter the number of vertices : \n");
    scanf("%d", &n);

    long h_adj[n * n];
    long* d_adj;
    hipMalloc((void**)&d_adj, n * n * sizeof(long*));

    setMax<<<1, dim3(n, n, 1)>>>(d_adj, n); 

    hipMemcpy(h_adj, d_adj, n * n * sizeof(long), hipMemcpyDeviceToHost);
    
    hipFree(d_adj);

    while(1){
        printf("Click 1 to enter edge and 0 to finish.\n");
        scanf("%d", &k);
        if(!k)
            break;
        int s, d, w;
        printf("Enter start and end of edge in 1-ordering : \n");
        scanf("%d %d", &s, &d);
        if(s == d){
            printf("Invalid edge.\n");
            continue;
        }
        if(s > n || s < 1 || d > n || d < 1){
            printf("Invalid edge.\n");
            continue;
        }
        printf("Enter edge weight : \n");
        scanf("%d", &w);
        if(w < 0){
            printf("Invalid edge weight.\n");
            continue;
        }
        int pos = ((s - 1) * n) + (d - 1);
        h_adj[pos] = w;
    }
    
    hipDeviceSynchronize();

    for(k = 0; k < n; k++){
        for(i = 0; i < n; i++){
            for(j = 0;j < n; j++){
                int pos1 = (i * n) + k;                                                 //adj[i][k]
                int pos2 = (k * n) + j;                                                 //adj[k][j]
                int pos3 = (i * n) + j;                                                 //adj[i][j]
                long s = (long)h_adj[pos1] + (long)h_adj[pos2];
                if(s < h_adj[pos3])
                    h_adj[pos3] = s;
            }
        }
    }

    for(i = 0;i < n; i++){
        for(j = 0;j < n; j++){
            int pos = (i * n) + j;    
            printf("%ld ", h_adj[pos]);
        }
        printf("\n");
    }

    return 0;
}
