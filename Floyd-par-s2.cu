
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<string.h>

__global__ void setMax(long* d_adj, int n){
    int x = threadIdx.x;
    int y = threadIdx.y;
    int pos = (x * n) + y;
    if(x == y)
        d_adj[pos] = 0;                                 //Diagonal elements
    else
        d_adj[pos] = __INT_MAX__;                       //Others
}

__global__ void compute(long *d_ad, int k, int n){
    int x = threadIdx.x;
    int y = threadIdx.y;
    int w_pos = (x * n) + y;
    int r_pos1 = (x * n) + k;
    int r_pos2 = (k * n) + y;
    long s = d_ad[r_pos1] + d_ad[r_pos2];

    __syncthreads();

    if(s < d_ad[w_pos])
        d_ad[w_pos] = s;

}

int main(int argc, char** argv){
    int i, j, k, n;
    printf("Enter the number of vertices : \n");
    scanf("%d", &n);

    long h_adj[n * n];
    long* d_adj;
    hipMalloc((void**)&d_adj, n * n * sizeof(long*));

    setMax<<<1, dim3(n, n, 1)>>>(d_adj, n); 

    hipMemcpy(h_adj, d_adj, n * n * sizeof(long), hipMemcpyDeviceToHost);
    
    hipFree(d_adj);

    while(1){
        printf("Click 1 to enter edge and 0 to finish.\n");
        scanf("%d", &k);
        if(!k)
            break;
        int s, d, w;
        printf("Enter start and end of edge in 1-ordering : \n");
        scanf("%d %d", &s, &d);
        if(s == d){
            printf("Invalid edge.\n");
            continue;
        }
        if(s > n || s < 1 || d > n || d < 1){
            printf("Invalid edge.\n");
            continue;
        }
        printf("Enter edge weight : \n");
        scanf("%d", &w);
        if(w < 0){
            printf("Invalid edge weight.\n");
            continue;
        }
        int pos = ((s - 1) * n) + (d - 1);
        h_adj[pos] = w;
    }
    
    hipDeviceSynchronize();

    long* d_ad;
    hipMalloc((void**)&d_ad, n * n * sizeof(long*));
    hipMemcpy(d_ad, h_adj, n * n * sizeof(long), hipMemcpyHostToDevice);


    for(k = 0; k < n; k++)
        compute<<<1, dim3(n, n, 1)>>>(d_ad, k, n);

    hipMemcpy(h_adj, d_ad, n * n * sizeof(long), hipMemcpyDeviceToHost);

    for(i = 0;i < n; i++){
        for(j = 0;j < n; j++){
            int pos = (i * n) + j;    
            printf("%ld ", h_adj[pos]);
        }
        printf("\n");
    }

    hipFree(d_ad);
    return 0;
}
