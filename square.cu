
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void square(float *d_out, float* d_in){
    int idx = threadIdx.x;
    float f = d_in[idx];
    d_out[idx] = f * f * f;
}

int main(int argc, char** argv){
    const int Ar_s = 96;
    const int Ar_b = Ar_s * sizeof(float);
 
    float h_in[Ar_s];
    int i;
    for(i = 0;i < Ar_s; i++)
        h_in[i] = float(i);

    float h_out[Ar_s];

    float *d_in, *d_out;
    hipMalloc((void **) &d_in, Ar_b);
    hipMalloc((void **) &d_out, Ar_b);

    hipMemcpy(d_in, h_in, Ar_b, hipMemcpyHostToDevice);

    square<<<1, Ar_s>>> (d_out, d_in);

    hipMemcpy(h_out, d_out, Ar_b, hipMemcpyDeviceToHost);

    for(i = 0; i < Ar_s; i++){
        printf("%f", h_out[i]);
        if(i%4 != 3)
            printf("\t");
        else
            printf("\n");
    }

    hipFree(d_in);
    hipFree(d_out);

    return 0;
}